#include "hip/hip_runtime.h"
#include "FlexibleFilter.hpp"

#define MAX_FILTERBANK_SIZE 10000
#define N_MAX_
#define N_MAX_CHANNELS 10
//We are just allocating max possible filterbank ahead of time
//whatever gets filled gets filled
//right now not expecting a filterbank larger than 10000 floats
//we'll see how that pans out
__device__ __constant__ float c_FilterBank[MAX_FILTERBANK_SIZE];

static __device__ float* getPtr(const DeviceMatrix mat, 
                         unsigned int r, unsigned int c)
{
    return mat.data + r*mat.pitch + c;
}

static __device__ float* getPtr(const DeviceMatrix3D mat, 
                         unsigned int t, unsigned int y, unsigned int x)
{
    return mat.data + t*mat.pitch_t + y*mat.pitch_y + x;
}

#define HIST_CACHE_SIZE 10
static const unsigned int MAX_HISTOGRAM_SIZE = 512;
static const unsigned int MAX_FOLDS = MAX_HISTOGRAM_SIZE / (BLOCK_8 * BLOCK_8);
__global__ void cell_histogram_kernel(DeviceMatrix3D input,
                                      DeviceMatrix3D output,
                                      const int cell_size,
                                      const int offset_y,
                                      const int offset_x,
                                      const int max_bin){

    const int out_y = blockIdx.y ;
    const int out_x = blockIdx.x ;

    const int n_threads = BLOCK_8 * BLOCK_8;
    const int thread_id =  threadIdx.y * BLOCK_8 + threadIdx.x ;

    __shared__ int   id_cache[BLOCK_8*BLOCK_8];
    __shared__ float wt_cache[BLOCK_8*BLOCK_8];

    const int base_y = offset_y + blockIdx.y * cell_size;
    const int base_x = offset_x + blockIdx.x * cell_size;

    const int cell_area = cell_size * cell_size;
    const int n_folds = (max_bin - 1) / n_threads + 1;

    float local_hist[MAX_FOLDS];

    //read the cell into the shared memory
    if ((threadIdx.y < cell_size) && (threadIdx.x < cell_size)){
        id_cache[threadIdx.y*cell_size + threadIdx.x] = 
            *getPtr(input, 0, base_y + threadIdx.y, base_x + threadIdx.x);

        wt_cache[threadIdx.y*cell_size + threadIdx.x] = 
            *getPtr(input, 1, base_y + threadIdx.y, base_x + threadIdx.x);
    }

    for (int i=0;i<MAX_FOLDS;i++){
        local_hist[i] = 0;
    }

    __syncthreads();

    //loop over the cell pixels and increment the histogram bin if the id matches
    for (int i=0; i<cell_area; i++){
        const int cur_id = id_cache[i];
        const float cur_wt = wt_cache[i];
        int hist_i = thread_id;
        for (int fi=0; fi<n_folds; fi++){
            if (cur_id == hist_i){
                local_hist[fi] += cur_wt; 
            }
            hist_i += n_threads;
        }
    }

    __syncthreads();

    //write out the histogram
    if ( (out_y < output.dim_t) && (out_x < output.dim_y) ){
        for (int fi=0; fi<n_folds; fi++){
            int hist_i = n_threads * fi + thread_id;
            if (hist_i < max_bin){
                *getPtr(output, out_y, out_x, hist_i) = local_hist[fi];
            }
        }
    }

    __syncthreads();
}

/* FOR COMPUTE CAPABILITY 2.0 AND ABOVE 
__global__ void cell_histogram_kernel(DeviceMatrix3D input,
                                      DeviceMatrix3D output,
                                      const int cell_size,
                                      const int offset_y,
                                      const int offset_x,
                                      const int max_bin){
  

    __shared__ float cell_histogram[MAX_HISTOGRAM_SIZE];

    int n_folds =  (max_bin - 1) / (BLOCK_SIZE * BLOCK_SIZE) + 1;

    for (int i=0; i<n_folds; i++){
        int hist_i = BLOCK_SIZE * BLOCK_SIZE * i + threadIdx.y * BLOCK_SIZE + threadIdx.x ;
        if (hist_i < max_bin){
            cell_histogram[hist_i] = 0;
        }
    }
    __syncthreads();

    int dim_folds = (cell_size - 1) / BLOCK_SIZE + 1;

    int base_y = offset_y + blockIdx.y * cell_size;
    int base_x = offset_x + blockIdx.x * cell_size;

    for (int i=0; i<dim_folds;i++){
        for (int j=0; j<dim_folds;j++){
            int thread_offset_y =  (BLOCK_SIZE * i) + threadIdx.y;
            int thread_offset_x =  (BLOCK_SIZE * j) + threadIdx.x;

            if ((thread_offset_y < cell_size) && (thread_offset_x < cell_size)){
                int id       = (int) *getPtr(input,0, base_y + thread_offset_y, base_x + thread_offset_x);
                float weight = *getPtr(input,1, base_y + thread_offset_y, base_x + thread_offset_x);

                //cell_histogram[id]+=weight;
                atomicAdd((float*)cell_histogram + id, weight);
            }
        }
    }

    __syncthreads();

    if ((threadIdx.x == 0)&&(threadIdx.y == 0)){
        for (int i=0;i<max_bin;i++){
            *getPtr(output, blockIdx.y, blockIdx.x, i) = 1;
        }
    }


    if ( (blockIdx.y < output.dim_t) && (blockIdx.x < output.dim_y) ){
        for (int i=0; i<n_folds;i++){
           int out_bin = threadIdx.y * (BLOCK_SIZE*BLOCK_SIZE) + threadIdx.x;
           if (out_bin < max_bin){
               *getPtr(output, blockIdx.y, blockIdx.x, out_bin) = 1;//cell_histogram[out_bin];
            }
        }
    }

    __syncthreads();
}
*/

__global__ void blockwise_filter_kernel(DeviceMatrix frame,
                                         DeviceMatrix3D output,
                                         const int frame_width, const int frame_height,
                                         const int apron_lo_y, const int apron_lo_x,
                                         const int apron_hi_y, const int apron_hi_x,
                                         const int dim_t, const int dim_y, const int dim_x,
                                         const int nchannels,
                                         const int optype)
{
    const int pix_y = blockIdx.y * (BLOCK_SIZE-dim_y+1) + threadIdx.y;
    const int pix_x = blockIdx.x * (BLOCK_SIZE-dim_x+1) + threadIdx.x;

    __shared__ float image_cache[BLOCK_SIZE][BLOCK_SIZE];

    __syncthreads();

    const int min_y = apron_hi_y;
    const int min_x = apron_hi_x;
    const int max_y = frame_height - apron_hi_y ;
    const int max_x = frame_width  - apron_hi_x ;

    for (int di=0; di<dim_t;di++){
        float tempval = 0;
        int fi_base = di * dim_y * dim_x * nchannels;
        for (int chan_id=0; chan_id<nchannels; chan_id++){
            int fi = chan_id + fi_base;

            image_cache[threadIdx.y][threadIdx.x] = 0;
            //load the particular channel
            if ( (pix_y >= 0) && (pix_y < frame_height) && (pix_x >= 0) && (pix_x < frame_width) ){
                image_cache[threadIdx.y][threadIdx.x] = *getPtr(frame, pix_y, pix_x * nchannels + chan_id);
            }

            __syncthreads();

            if (optype==FF_OPTYPE_EUCLIDEAN){
                for (int fyi=-apron_lo_y; fyi <= apron_hi_y; fyi++){
                    for (int fxi=-apron_lo_x; fxi <= apron_hi_x; fxi++){
                        float diff = image_cache[threadIdx.y+fyi][threadIdx.x+fxi] - c_FilterBank[fi];
                        tempval += diff*diff;
                        fi+=nchannels;
                    }
                }

            }
            else if (optype==FF_OPTYPE_COSINE){
                for (int fyi=-apron_lo_y; fyi <= apron_hi_y; fyi++){
                    for (int fxi=-apron_lo_x; fxi <= apron_hi_x; fxi++){
                        tempval += image_cache[threadIdx.y+fyi][threadIdx.x+fxi] * c_FilterBank[fi];
                        fi+=nchannels;
                    }
                }
            }
            __syncthreads();
        }

        if ( (pix_y >= min_y) && (pix_y < max_y) && (pix_x >= min_x) && (pix_x < max_x) ){
            if ( (threadIdx.y >= apron_lo_y) && (threadIdx.y < BLOCK_SIZE-apron_hi_y) && 
                 (threadIdx.x >= apron_lo_x) && (threadIdx.x < BLOCK_SIZE-apron_hi_x) ){
                *getPtr(output, pix_y, pix_x, di) = tempval;
            }
        }
        else if ( (pix_y >= 0) && (pix_y < frame_height) && (pix_x >= 0) && (pix_x < frame_width) ){
            *getPtr(output, pix_y, pix_x, di) = -1;
        }
    
    }

    __syncthreads();

}

/*
template <int FILTER_DIM>
__global__ void blockwise_distance_kernel(DeviceMatrix frame,
                                         DeviceMatrix3D output,
                                         const int frame_width, const int frame_height,
                                         const int dim_t,
                                         const int optype)
{
    const int pix_y = blockIdx.y * (BLOCK_SIZE-FILTER_DIM+1) + threadIdx.y;
    const int pix_x = blockIdx.x * (BLOCK_SIZE-FILTER_DIM+1) + threadIdx.x;

    const int out_pix_y = pix_y + FILTER_DIM / 2;
    const int out_pix_x = pix_x + FILTER_DIM / 2;

    const int out_pix_offset = out_pix_y*output.pitch_y + out_pix_x;

    __shared__ float image_cache[BLOCK_SIZE][BLOCK_SIZE];
    image_cache[threadIdx.y][threadIdx.x] = *(frame.data + pix_y*frame.pitch + pix_x);
    __syncthreads();

    float curval = -1e6;
    float curid = -1;
    float tempval;

    int fi=0;

    if ( (threadIdx.y < BLOCK_SIZE-FILTER_DIM+1) && (threadIdx.x < BLOCK_SIZE-FILTER_DIM+1) ){
        for (int filter_id=0; filter_id<dim_t; filter_id++){
            tempval = 0;

            if (optype==FF_OPTYPE_EUCLIDEAN) {
                for (int fyi=0; fyi<FILTER_DIM; fyi++){for (int fxi=0; fxi<FILTER_DIM; fxi++){
                    float diff = image_cache[threadIdx.y+fyi][threadIdx.x+fxi] - c_FilterBank[fi++];
                    tempval += diff * diff;
                }}
            }
            else { // optype==FF_OPTYPE_COSINE 
                for (int fyi=0; fyi<FILTER_DIM; fyi++){ for (int fxi=0; fxi<FILTER_DIM; fxi++){
                    tempval += c_FilterBank[fi++] * image_cache[threadIdx.y+fyi][threadIdx.x+fxi];
                }}
            }

            if (optype==FF_OPTYPE_EUCLIDEAN){
               if (tempval < curval){
                   curid = filter_id;
                   curval = tempval;
               }
            }
            else { //(optype==FF_OPTYPE_COSINE){
                if (abs(tempval) > curval){
                    curid = filter_id;
                    curval = abs(tempval);
                }
            }
        }
    }

    if ( (out_pix_y < frame_height) && (out_pix_x < frame_width) &&
         (threadIdx.y < BLOCK_SIZE - FILTER_DIM + 1) && (threadIdx.x < BLOCK_SIZE - FILTER_DIM + 1) ){
        *(output.data + out_pix_offset) = curid;
        *(output.data + output.pitch_t + out_pix_offset) = curval;
    }
}
*/

//this updates the filterbank saved in the constant device memory
int update_filter_bank_internal(float* new_filter, int filter_size){

    hipError_t cet;

    if (filter_size > MAX_FILTERBANK_SIZE){
        printf("ERROR: Filterbank too large\n");
        return 1;
    }
    else {
        //printf("Value in: %05f\n",new_filter[0]);
        cet = hipMemcpyToSymbol(HIP_SYMBOL(c_FilterBank), new_filter, sizeof(float) * filter_size,0, hipMemcpyHostToDevice);
        //printf("err: %d\n", cet);
        if (cet){
            printf("Some error happened while updating the filterbank\n");
            printf("Error code: %d\n",cet);
            return cet;
        }
        //printf("Value out:%04f\n",c_FilterBank[0]);
        return 0;
    }

}


#define BLOCK_MULT 2
template<int FILTER_DIM>
__global__ void blockwise_distance_kernel(DeviceMatrix frame,
                                         DeviceMatrix3D output,
                                         const int frame_width, const int frame_height,
                                         const int dim_t,
                                         const int optype)
{
    const int out_pix_y0 = blockIdx.y * (BLOCK_SIZE * BLOCK_MULT) + FILTER_DIM / 2;
    const int out_pix_x0 = blockIdx.x * (BLOCK_SIZE * BLOCK_MULT) + FILTER_DIM / 2;

    const int out_pix_y1 = min(out_pix_y0 + (BLOCK_SIZE * BLOCK_MULT),
                               frame_height - FILTER_DIM / 2);
    const int out_pix_x1 = min(out_pix_x0 + (BLOCK_SIZE * BLOCK_MULT),
                               frame_width - FILTER_DIM / 2);

    const int cache_size = BLOCK_SIZE * BLOCK_MULT + FILTER_DIM - 1;

    __shared__ float image_cache[cache_size][cache_size];

    int read_pix_y = out_pix_y0 - FILTER_DIM / 2 + threadIdx.y;
    int cache_ind_y = threadIdx.y;
    for (int ii=0; ii<BLOCK_MULT+1; ii++){
        int read_pix_x = out_pix_x0 - FILTER_DIM / 2 + threadIdx.x;
        int cache_ind_x = threadIdx.x;
        for (int jj=0; jj<BLOCK_MULT+1; jj++){
            if ((cache_ind_x < cache_size) && (cache_ind_y < cache_size)){
                image_cache[cache_ind_y][cache_ind_x] = *(frame.data + read_pix_y * frame.pitch + read_pix_x);
            }
            read_pix_x += BLOCK_SIZE;
            cache_ind_x += BLOCK_SIZE;
        }
        read_pix_y += BLOCK_SIZE;
        cache_ind_y += BLOCK_SIZE;
    }

    __syncthreads();

    int out_y = out_pix_y0 + threadIdx.y;
    for (int ii=0; ii<BLOCK_MULT; ii++){
        int out_x = out_pix_x0 + threadIdx.x;       
        for (int jj=0; jj<BLOCK_MULT; jj++){
            float curval = -1e6;
            float curid = -1;
            int fi = 0;
            if ((out_y < out_pix_y1) && (out_x < out_pix_x1)){           
                for (int filter_id=0; filter_id<dim_t; filter_id++){
                    float tempval = 0.0f;
                    int cyi = threadIdx.y + ii * BLOCK_SIZE;
                    for (int fyi=0; fyi<FILTER_DIM; fyi++){ 
                        int cxi = threadIdx.x + jj * BLOCK_SIZE;
                        for (int fxi=0; fxi<FILTER_DIM; fxi++){
                            tempval += c_FilterBank[fi++] * image_cache[cyi][cxi];
                            cxi++;
                        }
                        cyi++;
                    }
                    if (abs(tempval) > curval){
                        curid = filter_id;
                        curval = abs(tempval);
                    }
                }
           
                const int out_pix_offset = out_y * output.pitch_y + out_x;
                *(output.data + out_pix_offset) = curid;
                *(output.data + output.pitch_t + out_pix_offset) = curval;
            }
            out_x += BLOCK_SIZE;
        }
        out_y += BLOCK_SIZE;
    }

    __syncthreads();
}

void dist_filter2_d3(const DeviceMatrix* frame,
                  const int dim_t, const int nchannels,
                  DeviceMatrix3D* output,
                  const int optype)
{
    const int frame_width = int(frame->width);
    const int frame_height = int(frame->height);

    const int valid_region_h = frame_height - 3 + 1;
    const int valid_region_w = frame_width - 3 + 1;

    int grid_ry = valid_region_h / (BLOCK_SIZE * BLOCK_MULT) + 1;
    int grid_cx = valid_region_w / (BLOCK_SIZE * BLOCK_MULT) + 1;

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    dim3 dimGrid(grid_cx, grid_ry);

    blockwise_distance_kernel<3><<<dimGrid, dimBlock>>>(*frame,
                                                    *output,
                                                    frame_width, frame_height,
                                                    dim_t,
                                                    optype);
}




/**CUDA***/
void dist_filter2_d5(const DeviceMatrix* frame,
                  const int dim_t, const int nchannels,
                  DeviceMatrix3D* output,
                  const int optype)
{
    const int frame_width = int(frame->width);
    const int frame_height = int(frame->height);

    const int valid_region_h = frame_height - 5 + 1;
    const int valid_region_w = frame_width - 5 + 1;

    int grid_ry = valid_region_h / (BLOCK_SIZE * BLOCK_MULT) + 1;
    int grid_cx = valid_region_w / (BLOCK_SIZE * BLOCK_MULT) + 1;

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    dim3 dimGrid(grid_cx, grid_ry);

    blockwise_distance_kernel<5><<<dimGrid, dimBlock>>>(*frame,
                                                    *output,
                                                    frame_width, frame_height,
                                                    dim_t,
                                                    optype);

}
void dist_filter2_d7(const DeviceMatrix* frame,
                  const int dim_t, const int nchannels,
                  DeviceMatrix3D* output,
                  const int optype)
{
    const int frame_width = int(frame->width);
    const int frame_height = int(frame->height);

    const int valid_region_h = frame_height - 7 + 1;
    const int valid_region_w = frame_width - 7 + 1;

    int grid_ry = valid_region_h / (BLOCK_SIZE * BLOCK_MULT) + 1;
    int grid_cx = valid_region_w / (BLOCK_SIZE * BLOCK_MULT) + 1;

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    dim3 dimGrid(grid_cx, grid_ry);

    blockwise_distance_kernel<7><<<dimGrid, dimBlock>>>(*frame,
                                                    *output,
                                                    frame_width, frame_height,
                                                    dim_t,
                                                    optype);
}
/*
//new dist filter implementation
template<int FILTER_DIM>
void dist_filter2(const DeviceMatrix* frame,
                  const int dim_t, const int nchannels,
                  DeviceMatrix3D* output,
                  const int optype)
{
    const int frame_width = float(frame->width) / (nchannels);
    const int frame_height = float(frame->height);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    int grid_ry = (frame_height) / (dimBlock.y-FILTER_DIM+1) + 1;
    int grid_cx = (frame_width ) / (dimBlock.x-FILTER_DIM+1) + 1;
    dim3 dimGrid(grid_cx, grid_ry);

    blockwise_distance_kernel<FILTER_DIM><<<dimGrid, dimBlock>>>(*frame,
                                                    *output,
                                                    frame_width, frame_height,
                                                    dim_t,
                                                    optype);
   

}
*/

//new dist filter implementation
void dist_filter_noargmin(const DeviceMatrix* frame,
                  const int dim_t, const int dim_y, const int dim_x, const int nchannels,
                  DeviceMatrix3D* output,
                  const int optype)
{
    const int frame_width = frame->width / nchannels;
    const int frame_height = frame->height;

    const int apron_hi_y = dim_y / 2;
    const int apron_hi_x = dim_x / 2;

    const int apron_lo_y = dim_y - apron_hi_y - 1;
    const int apron_lo_x = dim_x - apron_hi_x - 1;

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    int grid_ry = (frame_height) / (dimBlock.y-dim_y+1) + 1;
    int grid_cx = (frame_width ) / (dimBlock.x-dim_x+1) + 1;
    dim3 dimGrid(grid_cx, grid_ry);

    blockwise_filter_kernel<<<dimGrid, dimBlock>>>(*frame,
                                                    *output,
                                                    frame_width, frame_height,
                                                    apron_lo_y, apron_lo_x,
                                                    apron_hi_y, apron_hi_x,
                                                    dim_t, dim_y, dim_x,
                                                    nchannels, optype);

}

void hist_all_cells(const DeviceMatrix3D* inds_and_weights,
                    DeviceMatrix3D* output,
                    const int cell_size,
                    const int offset_y,
                    const int offset_x,
                    const int max_bin){

    const int frame_height = inds_and_weights->dim_y;
    const int frame_width = inds_and_weights->dim_x;

    dim3 dimBlock(BLOCK_8, BLOCK_8);

    int grid_ry = (frame_height - offset_y) / cell_size + 1 ;
    int grid_cx = (frame_width  - offset_x) / cell_size + 1 ;
   
    dim3 dimGrid(grid_cx, grid_ry);
    
    cell_histogram_kernel<<<dimGrid, dimBlock>>>(*inds_and_weights,*output,cell_size,offset_y, offset_x, max_bin);

    hipDeviceSynchronize();
}


/*obsolete helper kernels. leaving for reference delete at will.
__global__ void copy_kernel(DeviceMatrix3D outmat,
                           DeviceMatrix newmat){
    const int pix_y = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    const int pix_x = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    if ( (pix_y < newmat.height) && (pix_x < newmat.width) ){
        float new_val = *getPtr(newmat, pix_y, pix_x);
        *getPtr(outmat, 0, pix_y, pix_x) = new_val;
        *getPtr(outmat, 1, pix_y, pix_x) = 0;
    }
    __syncthreads();
}

__global__ void min_kernel(DeviceMatrix3D outmat,
                           DeviceMatrix newmat,
                           const int new_label){

    const int pix_y = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    const int pix_x = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    if ( (pix_y < newmat.height) && (pix_x < newmat.width) ){
        float new_val = *getPtr(newmat, pix_y, pix_x);
        float out_val = *getPtr(outmat, 0, pix_y, pix_x);
        if (new_val < out_val){
            *getPtr(outmat, 0, pix_y, pix_x) = new_val;
            *getPtr(outmat, 1, pix_y, pix_x) = new_label;
        }
    }
    __syncthreads();

}

__global__ void max_kernel(DeviceMatrix3D outmat,
                           DeviceMatrix newmat,
                           const int new_label){

    const int pix_y = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    const int pix_x = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    if ( (pix_y < newmat.height) && (pix_x < newmat.width) ){
        float new_val = *getPtr(newmat, pix_y, pix_x);
        float out_val = *getPtr(outmat, 0, pix_y, pix_x);
        if (new_val > out_val){
            *getPtr(outmat, 0, pix_y, pix_x) = new_val;
            *getPtr(outmat, 1, pix_y, pix_x) = new_label;
        }
    }
    __syncthreads();

}

*/

/* obsolete euclid kernel called by the dist_filter
__global__ void blockwise_euclid_kernel(DeviceMatrix frame,
                                        DeviceMatrix3D filter_bank,
                                        DeviceMatrix output,
                                        const int filter_ind,
                                        const int start_y, const int start_x,
                                            const int end_y,   const int end_x)
{

    // Load the filter
    __shared__ float filter_cache[BLOCK_SIZE][BLOCK_SIZE];

    if ((threadIdx.y < filter_bank.dim_y) &&
        (threadIdx.x < filter_bank.dim_x) )
    {
        filter_cache[threadIdx.y][threadIdx.x] = *getPtr(filter_bank,
                                                         filter_ind,
                                                         threadIdx.y,
                                                         threadIdx.x);
    }

    __syncthreads();

    const int pix_y = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    const int pix_x = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    float out_val = 0;
    float temp_val = 0;
    if ((pix_y >= start_y)&&(pix_y <= end_y)&&
        (pix_x >= start_x)&&(pix_x <= end_x)){
        for (int i = -start_y; i<=start_y; i++) {
            for (int j = -start_x; j<=start_x; j++){
                temp_val = *getPtr(frame, pix_y+i, pix_x+j) - filter_cache[i+start_y][j+start_x];
                out_val += temp_val * temp_val;
                __syncthreads();
            }
        }
        *getPtr(output, pix_y, pix_x) = out_val;
    }

}
*/

/*obsolete cosine kernel called by dist_filter
__global__ void blockwise_cosine_kernel(DeviceMatrix frame,
                                        DeviceMatrix3D filter_bank,
                                        DeviceMatrix output,
                                        const int filter_ind,
                                        const int start_y, const int start_x,
                                        const int end_y,   const int end_x)
{

    // Load the filter
    __shared__ float filter_cache[BLOCK_SIZE][BLOCK_SIZE];

    if ((threadIdx.y < filter_bank.dim_y) &&
        (threadIdx.x < filter_bank.dim_x) )
    {
        filter_cache[threadIdx.y][threadIdx.x] = *getPtr(filter_bank,
                                                         filter_ind,
                                                         threadIdx.y,
                                                         threadIdx.x);
    }

    __syncthreads();

    const int pix_y = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    const int pix_x = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    float out_val = 0;
    //float sq_sum = 0;
    float pix_val = 0;
    if ((pix_y >= start_y)&&(pix_y <= end_y)&&
        (pix_x >= start_x)&&(pix_x <= end_x)){
        for (int i = -start_y; i<=start_y; i++) {
            for (int j = -start_x; j<=start_x; j++){
                pix_val = *getPtr(frame, pix_y+i, pix_x+j);
                out_val +=  pix_val * filter_cache[i+start_y][j+start_x];
                //sq_sum += pix_val * pix_val;
                __syncthreads();
            }
        }
        *getPtr(output, pix_y, pix_x) = out_val;// / sqrt(sq_sum);
    }

}
*/
/* obsoloete kernel for setting the edges of an image to -1
__global__ void clip_edges( const DeviceMatrix3D output,
                            const int start_y, const int start_x,
                            const int end_y, const int end_x)
{
    const int pix_y = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    const int pix_x = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    if ((pix_y < output.dim_y) && (pix_x < output.dim_x)){
        if ((pix_y < start_y) || (pix_x < start_x) ){
            *getPtr(output, 0, pix_y, pix_x) = -1;
            *getPtr(output, 1, pix_y, pix_x) = -1;
        }
        else if ((pix_y > end_y) || (pix_x > end_x) ) {
            *getPtr(output, 0, pix_y, pix_x) = -1;
            *getPtr(output, 1, pix_y, pix_x) = -1;
        }
    }

    __syncthreads();
}
*/



/* Obsolete implementation
void dist_filter( const DeviceMatrix* frame,
                  const int dim_t, const int dim_y, const int dim_x,
                  const DeviceMatrix3D* filter_bank,
                  DeviceMatrix3D* output,
                  const int optype)
{
    int half_filter_height = filter_bank->dim_y / 2;
    int half_filter_width  = filter_bank->dim_x / 2;

    int start_y = half_filter_height;
    int start_x = half_filter_width;

    int end_y = frame->height - half_filter_height - 1;
    int end_x = frame->width  - half_filter_width - 1;

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    int grid_ry = (frame->height-1) / dimBlock.y + 1;
    int grid_cx = (frame->width -1) / dimBlock.x + 1;

    dim3 dimGrid(grid_cx, grid_ry);

    DeviceMatrix::Ptr temp_result = makeDeviceMatrix(frame->height, frame->width);

    for (int filter_ind = 0; filter_ind < filter_bank->dim_t; filter_ind++){
        temp_result->zero();
        if (optype == FF_OPTYPE_EUCLIDEAN){
            blockwise_euclid_kernel<<<dimGrid, dimBlock>>>(*frame,
                                                       *filter_bank,
                                                       *temp_result,
                                                       filter_ind,
                                                       start_y, start_x,
                                                       end_y, end_x);
        }
        else if (optype == FF_OPTYPE_COSINE){
            blockwise_cosine_kernel<<<dimGrid, dimBlock>>>(*frame,
                                                       *filter_bank,
                                                       *temp_result,
                                                       filter_ind,
                                                       start_y, start_x,
                                                       end_y, end_x);
        }
        if (filter_ind == 0){
            copy_kernel<<<dimGrid, dimBlock>>>(*output, *temp_result);
        }
        else {
            if (optype == FF_OPTYPE_EUCLIDEAN){
                min_kernel<<<dimGrid, dimBlock>>>(*output, *temp_result, filter_ind);
            }
            else if (optype == FF_OPTYPE_COSINE){
                max_kernel<<<dimGrid, dimBlock>>>(*output, *temp_result, filter_ind);
            }
        }
    }

    clip_edges<<<dimGrid, dimBlock>>>(*output, start_y, start_x, end_y, end_x);
}
*/
